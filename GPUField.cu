#include "GPUField.cuh"
#include <cstdint>

template<typename T>
GPUField<T>::GPUField(unsigned int size, T initialValue) {
    m_size = size;
    m_hostData = new T[size];
    for (int i=0; i<size; i++) {
        m_hostData[i] = initialValue;
    }
    hipMalloc(&m_deviceData, size * sizeof(T));
}

template<typename T>
GPUField<T>::~GPUField() {
    delete[] m_hostData;
    hipFree(m_deviceData);
}

template<typename T>
void GPUField<T>::to_device() {
    hipMemcpy(m_deviceData, m_hostData, m_size * sizeof(T), hipMemcpyHostToDevice);
}

template<typename T>
void GPUField<T>::from_device() {
    hipMemcpy(m_hostData, m_deviceData, m_size * sizeof(T), hipMemcpyDeviceToHost);
}

template<typename T>
void GPUField<T>::from_device(T *hostData) {
    hipMemcpy(hostData, m_deviceData, m_size * sizeof(T), hipMemcpyDeviceToHost);
}

template<typename T>
unsigned int GPUField<T>::get_byte_size() {
    return m_size * sizeof(T);
}

template<typename T>
void GPUField<T>::set_host_data(T val) {
    for (int i=0; i<m_size; i++) {
        m_hostData[i] = val;
    }
}

template class GPUField<float>;
template class GPUField<uint8_t>;
template class GPUField<bool>;