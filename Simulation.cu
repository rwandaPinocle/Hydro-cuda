#include "hip/hip_runtime.h"
#include "Simulation.cuh"
#include <iostream>
#include <iomanip>


__device__ float sampleField(float *f, float x, float y, unsigned int w, unsigned int h, float xOffset, float yOffset) {
    // Calculate the coordinates of the sample location
    float newX = max(min(x - xOffset, (float) w), 0.0f);
    float newY = max(min(y - yOffset, (float) h), 0.0f);

    float xFrac = newX - (long)newX;
    float yFrac = newY - (long)newY;
    
    // Sample the smoke with bilinear interpolation
    float w11 = (1 - xFrac) * (1 - yFrac);
    float w12 = (1 - xFrac) * (    yFrac);
    float w21 = (    xFrac) * (1 - yFrac);
    float w22 = (    xFrac) * (    yFrac);

    return (
        w11 * f[(unsigned int)(((newY    ) * w) + (newX    ))] +
        w12 * f[(unsigned int)(((newY + 1) * w) + (newX    ))] +
        w21 * f[(unsigned int)(((newY    ) * w) + (newX + 1))] +
        w22 * f[(unsigned int)(((newY + 1) * w) + (newX + 1))] 
    );
}

__global__ void d_advect_vel(
    float *uField,
    float *vField,
    float *uNext,
    float *vNext,
    uint8_t *obstacles,
    unsigned int w,
    unsigned int h,
    float deltaT,
    float metersPerCell)
{
    int stride = gridDim.x * blockDim.x;
    int max_index = (w+1) * (h+1);

    for (int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < max_index; i+=stride) {
        int x = (i % (w + 1));
        int y = (i / (w + 1));
        
        //if (x == 0 || y == 0) {
        //    continue;
        //}

        // Add velocity to left side of the screen
        if (x == 0) {
            uField[i] = 0.02;
        }

        float u, v, newX, newY;

        // Advect u 
        u = uField[(x    ) + ((y    ) * (w + 1))];
        v = (
            vField[(x    ) + ((y    ) * (w    ))] +
            vField[(x    ) + ((y + 1) * (w    ))] + 
            vField[(x + 1) + ((y    ) * (w    ))] +
            vField[(x + 1) + ((y + 1) * (w    ))]
        ) / 4.0f;


        // Calculate the coordinates of the sample location
        newX = max(min(x - (u * deltaT / metersPerCell), (float) w), 0.0f);
        newY = max(min(y - (v * deltaT / metersPerCell), (float) h), 0.0f);

        float xFrac = newX - (long)newX;
        float yFrac = newY - (long)newY;
        
        // Sample uField
        float w11 = (1 - xFrac) * (1 - yFrac);
        float w12 = (1 - xFrac) * (    yFrac);
        float w21 = (    xFrac) * (1 - yFrac);
        float w22 = (    xFrac) * (    yFrac);

        if (x != w) {
            uNext[x + (y * (w + 1))] = (
                w11 * uField[(unsigned int)(((newY    ) * (w + 1)) + (newX    ))] +
                w12 * uField[(unsigned int)(((newY + 1) * (w + 1)) + (newX    ))] +
                w21 * uField[(unsigned int)(((newY    ) * (w + 1)) + (newX + 1))] +
                w22 * uField[(unsigned int)(((newY + 1) * (w + 1)) + (newX + 1))] 
            );
        }
        //uNext[i] = sampleField(uField, newX, newY, w, h, 0.0, 0.0);

        // Sample vField
        //newX = max(min(x - (u * deltaT / metersPerCell), (float) w), 0.0f);
        //newY = max(min(y - (v * deltaT / metersPerCell) - 0.5, (float) h), 0.0f);

        //xFrac = newX - (long)newX;
        //yFrac = newY - (long)newY;

        //w11 = (1 - xFrac) * (1 - yFrac);
        //w12 = (1 - xFrac) * (    yFrac);
        //w21 = (    xFrac) * (1 - yFrac);
        //w22 = (    xFrac) * (    yFrac);

        //vNext[i] = (
        //    w11 * vField[(unsigned int)(((newY    ) * w) + (newX    ))] +
        //    w12 * vField[(unsigned int)(((newY + 1) * w) + (newX    ))] +
        //    w21 * vField[(unsigned int)(((newY    ) * w) + (newX + 1))] +
        //    w22 * vField[(unsigned int)(((newY + 1) * w) + (newX + 1))] 
        //);
    }
}

__global__ void d_advect_smoke(
    float *smoke,
    float *smokeNext,
    float *uField,
    float *vField,
    uint8_t *obstacles,
    unsigned int w,
    unsigned int h,
    float deltaT,
    float metersPerCell)
{
    int stride = gridDim.x * blockDim.x;
    int max_index = w * h;

    for (int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < max_index; i+=stride) {
        int x = i % w;
        int y = i / w;

        // Advect smoke
        // Find the velocities at x, y
        float u = (uField[(x    ) + ((y    ) * (w + 1))] + uField[(x + 1) + ((y    ) * (w + 1))]) / 2.0;
        float v = (vField[(x    ) + ((y    ) * (w    ))] + vField[(x    ) + ((y + 1) * (w    ))]) / 2.0;

        // Add smoke to left side of the screen
        if ((x > -1) && (x < 2) && (y < h) && (y > -1)) {
            smoke[i] = 1.0;
        }

        // Calculate the coordinates of the sample location
        float newX = max(min(x - (u * deltaT / metersPerCell), (float) w), 0.0f);
        float newY = max(min(y - (v * deltaT / metersPerCell), (float) h), 0.0f);

        smokeNext[i] = sampleField(smoke, newX, newY, w, h, 0.0f, 0.0f);
    }
}

__global__ void d_render_texture(uint8_t *pixels, float *smoke, float *uField, float *vField, unsigned int width, unsigned int height) {
    int stride = gridDim.x * blockDim.x;
    int max_index = width * height;

    for (int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < max_index; i+=stride) {
        // Clamp the smoke value to 0-255
        //int pixel_value = static_cast<int>(uField[i * width/(width + 1)] * 25500 * 2);
        int pixel_value = static_cast<int>(smoke[i] * 255);
        if (pixel_value > 255) {
            pixel_value = 255;
        } else if (pixel_value < 0) {
            pixel_value = 0;
        }
        pixels[(4*i) + 1] = pixel_value;
    }
}

Simulation::Simulation(unsigned int width, unsigned int height, float dt) {
    m_width = width;
    m_height = height;
    m_u =         new GPUField<float>(  (width + 1) *  height     , 0.05);
    m_v =         new GPUField<float>(   width      * (height + 1), 0.00);
    m_uNext =     new GPUField<float>(  (width + 1) *  height     );
    m_vNext =     new GPUField<float>(   width      * (height + 1));
    m_smoke =     new GPUField<float>(   width      *  height     );
    m_smokeNext = new GPUField<float>(   width      *  height     );
    m_obstacles = new GPUField<uint8_t>( width      *  height     );
    m_pixels =    new GPUField<uint8_t>(4 * width   * height);
}

Simulation::~Simulation() {
    delete m_u;
    delete m_v;
    delete m_uNext;
    delete m_vNext;
    delete m_smoke;
    delete m_obstacles;
    delete m_pixels;
}

void Simulation::to_device(){
    m_u->to_device();
    m_v->to_device();
    m_uNext->to_device();
    m_vNext->to_device();
    m_smoke->to_device();
    m_smokeNext->to_device();
    m_obstacles->to_device();

    // d_pixels only goes from device to host
    //d_pixels.to_device();
}

void Simulation::from_device(){
    m_u->from_device();
    m_v->from_device();
    m_uNext->from_device();
    m_vNext->from_device();
    m_smoke->from_device();
    m_smokeNext->from_device();
    m_obstacles->from_device();
}

void Simulation::step() {
    //project();
    //advect_velocity();

    //m_u->m_hostData[0] = 0.11;
    //m_u->m_hostData[1] = 0.22;
    this->to_device();   
    //printf("u: %f\n", m_u->m_hostData[1]);

    d_advect_smoke<<<1000, 256>>>(
        m_smoke->m_deviceData,
        m_smokeNext->m_deviceData,
        m_u->m_deviceData,
        m_v->m_deviceData,
        m_obstacles->m_deviceData,
        m_width,
        m_height,
        0.0001,
        0.0001);

    d_advect_vel<<<1000, 256>>>(
        m_u->m_deviceData,
        m_v->m_deviceData,
        m_uNext->m_deviceData,
        m_vNext->m_deviceData,
        m_obstacles->m_deviceData,
        m_width,
        m_height,
        0.0001,
        0.0001);

    this->from_device();

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }

    GPUField<float> *temp;

    temp = m_u;
    m_u = m_uNext;
    m_uNext = temp;

    temp = m_v;
    m_v = m_vNext;
    m_vNext = temp;
    
    temp = m_smoke;
    m_smoke = m_smokeNext;
    m_smokeNext = temp;
}

void Simulation::render_texture(uint8_t *pixels) {
    d_render_texture<<<1000, 256>>>(m_pixels->m_deviceData, m_smoke->m_deviceData, m_u->m_deviceData, m_v->m_deviceData, m_width, m_height);
    m_pixels->from_device(pixels);
}
