#include "hip/hip_runtime.h"
#include "Simulation.cuh"
#include <iostream>
#include <iomanip>
#include <cmath>

void swapFields(GPUField<float> **a, GPUField<float> **b) {
    GPUField<float> *temp = *a;
    *a = *b;
    *b = temp;
}

__device__ float sampleField(
    float *f,
    float x,
    float y,
    unsigned int w,
    unsigned int h,
    float xOffset,
    float yOffset
) {
    // Calculate the coordinates of the sample location
    float newX = max(min(x, (float) w-1), 0.0f);
    float newY = max(min(y, (float) h-1), 0.0f);
    newX = x;
    newY = y;

    float xFrac = newX - (long)newX;
    float yFrac = newY - (long)newY;
    
    // Sample the field at the new location with bilinear interpolation
    float w11 = (1 - xFrac) * (1 - yFrac);
    float w12 = (1 - xFrac) * (    yFrac);
    float w21 = (    xFrac) * (1 - yFrac);
    float w22 = (    xFrac) * (    yFrac);

    unsigned int x1 = newX;
    unsigned int y1 = newY;
    unsigned int x2 = min((unsigned int) (newX + 1), w-1);
    unsigned int y2 = min((unsigned int) (newY + 1), h-1);

    return (
        w11 * f[x1 + y1*w] +
        w12 * f[x1 + y2*w] +
        w21 * f[x2 + y1*w] +
        w22 * f[x2 + y2*w] 
    );
}

__global__ void d_advect_vel(
    float *uField,
    float *vField,
    float *uNext,
    float *vNext,
    float *obstacles,
    unsigned int w,
    unsigned int h,
    float deltaT,
    float metersPerCell)
{
    int stride = gridDim.x * blockDim.x;
    int max_index = (w+1) * (h+1);

    for (int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < max_index; i+=stride) {
        int x = (i % (w + 1));
        int y = (i / (w + 1));

        float u, v, newX, newY;

        if (x != w) {
            // Advect u 
            u = uField[(x    ) + ((y    ) * (w + 1))];
            v = (
                vField[(x    ) + ((y    ) * (w    ))] +
                vField[(x    ) + ((y + 1) * (w    ))] + 
                vField[(x + 1) + ((y    ) * (w    ))] +
                vField[(x + 1) + ((y + 1) * (w    ))]
            ) / 4.0f;


            // Calculate the coordinates of the sample location
            newX = max(min(x - (u * deltaT / metersPerCell), (float) w), 0.0f);
            newY = max(min(y - (v * deltaT / metersPerCell), (float) h), 0.0f);

            uNext[x + (y * (w + 1))] = sampleField(uField, newX, newY, w + 1, h + 1, 0.0, 0.0);
        }

        if (y != h) {
            // Advect v 
            u = (
                uField[(x    ) + ((y    ) * (w + 1))] +
                uField[(x + 1) + ((y    ) * (w + 1))] +
                uField[(x    ) + ((y + 1) * (w + 1))] +
                uField[(x + 1) + ((y + 1) * (w + 1))] 
            ) / 4.0f;
            v = vField[(x    ) + ((y    ) * (w    ))];

            // Calculate the coordinates of the sample location
            newX = max(min(x - (u * deltaT / metersPerCell), (float) w), 0.0f);
            newY = max(min(y - (v * deltaT / metersPerCell), (float) h), 0.0f);

            vNext[x + (y * w)] = sampleField(vField, newX, newY, w, h + 1, 0.0, 0.0);
        }
    }
}

__global__ void d_advect_smoke(
    float *smoke,
    float *smokeNext,
    float *uField,
    float *vField,
    float *obstacles,
    unsigned int w,
    unsigned int h,
    float deltaT,
    float metersPerCell)
{
    int stride = gridDim.x * blockDim.x;
    int max_index = w * h;

    for (int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < max_index; i+=stride) {
        int x = i % w;
        int y = i / w;

        // Add smoke to the screen
        if (x == 1 && (y % 10 < 2)) {
            smoke[i] = 1.0;
            smoke[i+1] = 1.0;
            smoke[i+2] = 1.0;
            smoke[i+3] = 1.0;
            smoke[i+4] = 1.0;
        }

        // Find the velocities at x, y
        float u = (uField[(x    ) + ((y    ) * (w + 1))] + uField[(x + 1) + ((y    ) * (w + 1))]) / 2.0;
        float v = (vField[(x    ) + ((y    ) * (w    ))] + vField[(x    ) + ((y + 1) * (w    ))]) / 2.0;

        // Calculate the coordinates of the sample location
        float newX = max(min(x - (u * deltaT / metersPerCell), (float) w), 0.0f);
        float newY = max(min(y - (v * deltaT / metersPerCell), (float) h), 0.0f);

        smokeNext[i] = sampleField(smoke, newX, newY, w, h, 0.0f, 0.0f);
    }
}

__host__ __device__ void project_cell(
    float *u,
    float *v,
    float *obs,
    unsigned int w,
    unsigned int h,
    unsigned int x,
    unsigned int y,
    float overrelaxation
) {
    unsigned int obsIdxX0 = (x - 1) + ((y    ) * w);
    unsigned int obsIdxX1 = (x + 1) + ((y    ) * w);
    unsigned int obsIdxY0 = (x    ) + ((y - 1) * w);
    unsigned int obsIdxY1 = (x    ) + ((y + 1) * w);

    unsigned int uIdx0 =  x      + ( y      * (w + 1));
    unsigned int uIdx1 = (x + 1) + ( y      * (w + 1));

    unsigned int vIdx0 =  x      + ( y      *  w     );
    unsigned int vIdx1 =  x      + ((y + 1) *  w     );

    float obsCount = 4.0f - (
        obs[obsIdxX0] +
        obs[obsIdxX1] +
        obs[obsIdxY0] +
        obs[obsIdxY1]
    );
    if (obsCount == 4.0) {
        return;
    }

    float divergence = (
        - u[uIdx0] + u[uIdx1]
        - v[vIdx0] + v[vIdx1]
    ) / (4 - obsCount);

    float p = -divergence/(4 - obsCount);
    p *= overrelaxation;

    u[uIdx0] -= (obs[obsIdxX0] * p);
    u[uIdx1] += (obs[obsIdxX1] * p);
    v[vIdx0] -= (obs[obsIdxY0] * p);
    v[vIdx1] += (obs[obsIdxY1] * p);
}

__global__ void d_project(
    float *u,
    float *v,
    float *obs,
    unsigned int w,
    unsigned int h
) {
    int stride = gridDim.x * blockDim.x;
    int max_index = (w-2) * (h-2);
    float overrelaxation = 1.9;

    for (int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < max_index; i+=stride) {
        int x = (i % (w-2)) + 1;
        int y = (i / (w-2)) + 1;

        project_cell(u, v, obs, w, h, x, y, overrelaxation);
    }
}

void h_project(float *u,
    float *v,
    float *obs,
    unsigned int w,
    unsigned int h
) {
    int max_index = (w-2) * (h-2);
    float overrelaxation = 1.9;

    for (int i=0; i<max_index; i++) {
        int x = (i % (w-2)) + 1;
        int y = (i / (w-2)) + 1;
    
        project_cell(u, v, obs, w, h, x, y, overrelaxation);
    }
}

__global__ void d_render_texture(
    uint8_t *pixels,
    float *smoke,
    float *uField,
    float *vField,
    float *obs,
    unsigned int width,
    unsigned int height
) {
    int stride = gridDim.x * blockDim.x;
    int max_index = width * height;

    for (int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < max_index; i+=stride) {
        // Clamp the smoke value to 0-255
        int green = static_cast<int>(smoke[i] * 255);
        if (green > 255) {
            green = 255;
        } else if (green < 0) {
            green = 0;
        }
        pixels[(4*i) + 1] = green;

        int red = static_cast<int>((1.0 - obs[i]) * 255);
        pixels[(4*i) + 2] = red;
    }
}

Simulation::Simulation(unsigned int width, unsigned int height, float dt) {
    m_width = width;
    m_height = height;
    m_u =         new GPUField<float>(  (width + 1) *  height     , 0.0);
    m_v =         new GPUField<float>(   width      * (height + 1), 0.0);
    m_uNext =     new GPUField<float>(  (width + 1) *  height     );
    m_vNext =     new GPUField<float>(   width      * (height + 1));
    m_smoke =     new GPUField<float>(   width      *  height     );
    m_smokeNext = new GPUField<float>(   width      *  height     );
    m_obstacles = new GPUField<float>(   width      *  height     , 1.0);
    m_pixels =    new GPUField<uint8_t>(4 * width   * height);

    // Add obstacles
    float radius = 7;
    for (int i=0; i<width; i++) {
        for (int j=0; j<height; j++) {
            if (i == 0 || j == 0 || j == height - 1) {
                m_obstacles->m_hostData[i + j * width] = 0.0;
            }  
            if (pow((float)i-((float)width/5), 2) + pow((float)j-((float)height/2), 2) < pow(radius, 2)) {
                m_obstacles->m_hostData[i + j * width] = 0.0;
            }
        }
    }
}

Simulation::~Simulation() {
    delete m_u;
    delete m_v;
    delete m_uNext;
    delete m_vNext;
    delete m_smoke;
    delete m_obstacles;
    delete m_pixels;
}

void Simulation::to_device(){
    m_u->to_device();
    m_v->to_device();
    m_uNext->to_device();
    m_vNext->to_device();
    m_smoke->to_device();
    m_smokeNext->to_device();
    m_obstacles->to_device();

    // d_pixels is excluded because it only goes from device to host
}

void Simulation::from_device(){
    m_u->from_device();
    m_v->from_device();
    m_uNext->from_device();
    m_vNext->from_device();
    m_smoke->from_device();
    m_smokeNext->from_device();
    m_obstacles->from_device();
}


void Simulation::step() {
    // Add velocity to left side of the screen
    for (int y=0; y<m_height; y++) {
        m_u->m_hostData[(y * (m_width + 1)) + 1] = 15.0;
    }

    this->to_device();   

    const bool useGPU = true;
    int iterations = 100;
    for (int i=0; i<iterations; i++) {
        if (useGPU) {
            d_project<<<1, 256>>>(
                m_u->m_deviceData,
                m_v->m_deviceData,
                m_obstacles->m_deviceData,
                m_width,
                m_height
            );
        } else {
            h_project(
                m_u->m_hostData,
                m_v->m_hostData,
                m_obstacles->m_hostData,
                m_width,
                m_height
            );
        }
    }
    if (!useGPU) {
        this->to_device();
    }

    d_advect_vel<<<1000, 256>>>(
        m_u->m_deviceData,
        m_v->m_deviceData,
        m_uNext->m_deviceData,
        m_vNext->m_deviceData,
        m_obstacles->m_deviceData,
        m_width,
        m_height,
        0.0001,
        0.001);

    d_advect_smoke<<<1000, 256>>>(
        m_smoke->m_deviceData,
        m_smokeNext->m_deviceData,
        m_u->m_deviceData,
        m_v->m_deviceData,
        m_obstacles->m_deviceData,
        m_width,
        m_height,
        0.0001,
        0.001);

    this->from_device();

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }

    swapFields(&m_u, &m_uNext);
    swapFields(&m_v, &m_vNext);
    swapFields(&m_smoke, &m_smokeNext);
}

void Simulation::render_texture(uint8_t *pixels) {
    d_render_texture<<<1000, 256>>>(
        m_pixels->m_deviceData,
        m_smoke->m_deviceData,
        m_u->m_deviceData,
        m_v->m_deviceData,
        m_obstacles->m_deviceData,
        m_width,
        m_height
    );
    m_pixels->from_device(pixels);
}
