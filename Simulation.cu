#include "hip/hip_runtime.h"
#include "Simulation.cuh"
#include <iostream>
#include <iomanip>
#include <cmath>


__device__ float sampleField(float *f, float x, float y, unsigned int w, unsigned int h, float xOffset, float yOffset) {
    // Calculate the coordinates of the sample location
    float newX = max(min(x, (float) w-1), 0.0f);
    float newY = max(min(y, (float) h-1), 0.0f);
    newX = x;
    newY = y;

    float xFrac = newX - (long)newX;
    float yFrac = newY - (long)newY;
    
    // Sample the smoke with bilinear interpolation
    float w11 = (1 - xFrac) * (1 - yFrac);
    float w12 = (1 - xFrac) * (    yFrac);
    float w21 = (    xFrac) * (1 - yFrac);
    float w22 = (    xFrac) * (    yFrac);

    unsigned int x1 = newX;
    unsigned int y1 = newY;
    unsigned int x2 = min((unsigned int) (newX + 1), w-1);
    unsigned int y2 = min((unsigned int) (newY + 1), h-1);

    return (
        w11 * f[x1 + y1*w] +
        w12 * f[x1 + y2*w] +
        w21 * f[x2 + y1*w] +
        w22 * f[x2 + y2*w] 
    );
}

__global__ void d_advect_vel(
    float *uField,
    float *vField,
    float *uNext,
    float *vNext,
    float *obstacles,
    unsigned int w,
    unsigned int h,
    float deltaT,
    float metersPerCell)
{
    int stride = gridDim.x * blockDim.x;
    int max_index = (w+1) * (h+1);

    for (int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < max_index; i+=stride) {
        int x = (i % (w + 1));
        int y = (i / (w + 1));
        
        //if (x == 0 || y == 0) {
        //    continue;
        //}

        // Add velocity to left side of the screen
        //if (x == 0) {
        //    uField[i] = 0.001;
        //}

        float u, v, newX, newY;

        if (x != w) {
            // Advect u 
            u = uField[(x    ) + ((y    ) * (w + 1))];
            v = (
                vField[(x    ) + ((y    ) * (w    ))] +
                vField[(x    ) + ((y + 1) * (w    ))] + 
                vField[(x + 1) + ((y    ) * (w    ))] +
                vField[(x + 1) + ((y + 1) * (w    ))]
            ) / 4.0f;


            // Calculate the coordinates of the sample location
            newX = max(min(x - (u * deltaT / metersPerCell), (float) w), 0.0f);
            newY = max(min(y - (v * deltaT / metersPerCell), (float) h), 0.0f);

            uNext[x + (y * (w + 1))] = sampleField(uField, newX, newY, w + 1, h + 1, 0.0, 0.0);
        }

        if (y != h) {
            // Advect v 
            u = (
                uField[(x    ) + ((y    ) * (w + 1))] +
                uField[(x + 1) + ((y    ) * (w + 1))] +
                uField[(x    ) + ((y + 1) * (w + 1))] +
                uField[(x + 1) + ((y + 1) * (w + 1))] 
            ) / 4.0f;
            v = vField[(x    ) + ((y    ) * (w    ))];

            // Calculate the coordinates of the sample location
            newX = max(min(x - (u * deltaT / metersPerCell), (float) w), 0.0f);
            newY = max(min(y - (v * deltaT / metersPerCell), (float) h), 0.0f);

            vNext[x + (y * w)] = sampleField(vField, newX, newY, w, h + 1, 0.0, 0.0);
        }
    }
}

__global__ void d_advect_smoke(
    float *smoke,
    float *smokeNext,
    float *uField,
    float *vField,
    float *obstacles,
    unsigned int w,
    unsigned int h,
    float deltaT,
    float metersPerCell)
{
    int stride = gridDim.x * blockDim.x;
    int max_index = w * h;

    for (int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < max_index; i+=stride) {
        int x = i % w;
        int y = i / w;

        // Add smoke to the screen
        if (x == 1 && (y % 10 < 2)) {
            smoke[i] = 1.0;
            smoke[i+1] = 1.0;
            smoke[i+2] = 1.0;
            smoke[i+3] = 1.0;
            smoke[i+4] = 1.0;
        }
        //const int radius = 4;
        //if (x > ((w/2) - radius) && x < ((w/2) + radius) && y > ((h/2) - radius) && y < ((h/2) + radius)) {
        //    smoke[i] = 1.0;
        //}
        
        /*
        if (x == 1 || x == 0) {
            smoke[i] = 1.0;
        }
        */

        // Advect smoke
        // Find the velocities at x, y
        float u = (uField[(x    ) + ((y    ) * (w + 1))] + uField[(x + 1) + ((y    ) * (w + 1))]) / 2.0;
        float v = (vField[(x    ) + ((y    ) * (w    ))] + vField[(x    ) + ((y + 1) * (w    ))]) / 2.0;


        // Calculate the coordinates of the sample location
        float newX = max(min(x - (u * deltaT / metersPerCell), (float) w), 0.0f);
        float newY = max(min(y - (v * deltaT / metersPerCell), (float) h), 0.0f);

        smokeNext[i] = sampleField(smoke, newX, newY, w, h, 0.0f, 0.0f);
        //printf("u: %f\tv: %f\tx: %d\ty: %d\tnewX: %f\tnewY: %f\tsmoke[i]: %f\t smokeNext[i]: %f\n",
        //    u, v, x, y, newX, newY, smoke[i], smokeNext[i]);

    }
}

__host__ __device__ void project_cell(float *u, float *v, float *obs, unsigned int w, unsigned int h, unsigned int x, unsigned int y, float overrelaxation) {
    unsigned int obsIdxX0 = (x - 1) + ((y    ) * w);
    unsigned int obsIdxX1 = (x + 1) + ((y    ) * w);
    unsigned int obsIdxY0 = (x    ) + ((y - 1) * w);
    unsigned int obsIdxY1 = (x    ) + ((y + 1) * w);

    unsigned int uIdx0 =  x      + ( y      * (w + 1));
    unsigned int uIdx1 = (x + 1) + ( y      * (w + 1));

    unsigned int vIdx0 =  x      + ( y      *  w     );
    unsigned int vIdx1 =  x      + ((y + 1) *  w     );

    float obsCount = 4.0f - (
        obs[obsIdxX0] +
        obs[obsIdxX1] +
        obs[obsIdxY0] +
        obs[obsIdxY1]
    );
    if (obsCount == 4.0) {
        return;
    }

    float divergence = (
        - u[uIdx0] + u[uIdx1]
        - v[vIdx0] + v[vIdx1]
    ) / (4 - obsCount);

    float p = -divergence/(4 - obsCount);
    p *= overrelaxation;

    u[uIdx0] -= (obs[obsIdxX0] * p);
    u[uIdx1] += (obs[obsIdxX1] * p);
    v[vIdx0] -= (obs[obsIdxY0] * p);
    v[vIdx1] += (obs[obsIdxY1] * p);
}

__global__ void d_project(float *u, float *v, float *obs, unsigned int w, unsigned int h, unsigned int parity) {
    int stride = gridDim.x * blockDim.x;
    //printf("stride: %d\ti: %d\tblockDim.x: %d\tblockIdx.x: %d\tthreadIdx.x: %d\n", stride, (blockDim.x * blockIdx.x) + threadIdx.x, blockDim.x, blockIdx.x, threadIdx.x);
    int max_index = (w-2) * (h-2);
    float overrelaxation = 1.9;

    for (int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < max_index; i+=stride) {
        //printf("d_project i: %02d\n", i);
        int x = (i % (w-2)) + 1;
        int y = (i / (w-2)) + 1;

        if (x % 2 == parity && y % 2 == parity) {
            continue;
        }
        project_cell(u, v, obs, w, h, x, y, overrelaxation);
    }
}

void h_project(float *u, float *v, float *obs, unsigned int w, unsigned int h, unsigned int parity) {
    int max_index = (w-2) * (h-2);
    float overrelaxation = 1.9;

    for (int i=0; i<max_index; i++) {
        //printf("h_project i: %02d\n", i);
        int x = (i % (w-2)) + 1;
        int y = (i / (w-2)) + 1;
    
        if (x % 2 == parity && y % 2 == parity) {
            continue;
        }
        project_cell(u, v, obs, w, h, x, y, overrelaxation);
    }
}

__global__ void d_render_texture(
    uint8_t *pixels,
    float *smoke,
    float *uField,
    float *vField,
    float *obs,
    unsigned int width,
    unsigned int height
) {
    int stride = gridDim.x * blockDim.x;
    int max_index = width * height;

    for (int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < max_index; i+=stride) {
        // Clamp the smoke value to 0-255
        //int pixel_value = static_cast<int>(uField[i * width/(width + 1)] * 25500 * 2);
        int green = static_cast<int>(smoke[i] * 255);
        if (green > 255) {
            green = 255;
        } else if (green < 0) {
            green = 0;
        }
        pixels[(4*i) + 1] = green;

        int red = static_cast<int>((1.0 - obs[i]) * 255);
        pixels[(4*i) + 2] = red;
    }
}

Simulation::Simulation(unsigned int width, unsigned int height, float dt) {
    m_width = width;
    m_height = height;
    m_u =         new GPUField<float>(  (width + 1) *  height     , 0.0);
    m_v =         new GPUField<float>(   width      * (height + 1), 0.0);
    m_uNext =     new GPUField<float>(  (width + 1) *  height     );
    m_vNext =     new GPUField<float>(   width      * (height + 1));
    m_smoke =     new GPUField<float>(   width      *  height     );
    m_smokeNext = new GPUField<float>(   width      *  height     );
    m_obstacles = new GPUField<float>(   width      *  height     , 1.0);
    m_pixels =    new GPUField<uint8_t>(4 * width   * height);

    // Add obstacles
    float radius = 20;
    for (int i=0; i<width; i++) {
        for (int j=0; j<height; j++) {
            if (i == 0 || j == 0 || j == height - 1) {
                m_obstacles->m_hostData[i + j * width] = 0.0;
            }  
            if (pow((float)i-((float)width/5), 2) + pow((float)j-((float)height/2), 2) < pow(radius, 2)) {
                m_obstacles->m_hostData[i + j * width] = 0.0;
            }
        }
    }

}

Simulation::~Simulation() {
    delete m_u;
    delete m_v;
    delete m_uNext;
    delete m_vNext;
    delete m_smoke;
    delete m_obstacles;
    delete m_pixels;
}

void Simulation::to_device(){
    m_u->to_device();
    m_v->to_device();
    m_uNext->to_device();
    m_vNext->to_device();
    m_smoke->to_device();
    m_smokeNext->to_device();
    m_obstacles->to_device();

    // d_pixels only goes from device to host
    //d_pixels.to_device();
}

void Simulation::from_device(){
    m_u->from_device();
    m_v->from_device();
    m_uNext->from_device();
    m_vNext->from_device();
    m_smoke->from_device();
    m_smokeNext->from_device();
    m_obstacles->from_device();
}

void Simulation::step() {
    //project();
    //advect_velocity();

    //m_u->m_hostData[0] = 0.11;
    //m_u->m_hostData[1] = 0.22;
    //printf("u: %f\n", m_u->m_hostData[1]);

    // Add velocity to left side of the screen
    for (int y=0; y<m_height; y++) {
        m_u->m_hostData[(y * (m_width + 1)) + 1] = 15.0;
        //m_v->m_hostData[(i * m_width) + 10] = 0.01;
    }

    this->to_device();   

    int iterations = 100;
    for (int i=0; i<iterations; i++) {
        d_project<<<1, 256>>>(
            m_u->m_deviceData,
            m_v->m_deviceData,
            m_obstacles->m_deviceData,
            m_width,
            m_height,
            0
        );
        d_project<<<1, 256>>>(
            m_u->m_deviceData,
            m_v->m_deviceData,
            m_obstacles->m_deviceData,
            m_width,
            m_height,
            1
        );
        //h_project(
        //    m_u->m_hostData,
        //    m_v->m_hostData,
        //    m_obstacles->m_hostData,
        //    m_width,
        //    m_height,
        //    0
        //);

        //h_project(
        //    m_u->m_hostData,
        //    m_v->m_hostData,
        //    m_obstacles->m_hostData,
        //    m_width,
        //    m_height,
        //    1
        //);
        
        /*
        GPUField<float> *temp;
        temp = m_u;
        m_u = m_uNext;
        m_uNext = temp;

        temp = m_v;
        m_v = m_vNext;
        m_vNext = temp;
        */
    }
    //this->to_device();

    d_advect_vel<<<1000, 256>>>(
        m_u->m_deviceData,
        m_v->m_deviceData,
        m_uNext->m_deviceData,
        m_vNext->m_deviceData,
        m_obstacles->m_deviceData,
        m_width,
        m_height,
        0.0001,
        0.001);

    d_advect_smoke<<<1000, 256>>>(
        m_smoke->m_deviceData,
        m_smokeNext->m_deviceData,
        m_u->m_deviceData,
        m_v->m_deviceData,
        m_obstacles->m_deviceData,
        m_width,
        m_height,
        0.0001,
        0.001);

    this->from_device();

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }

    GPUField<float> *temp;

    temp = m_u;
    m_u = m_uNext;
    m_uNext = temp;

    temp = m_v;
    m_v = m_vNext;
    m_vNext = temp;
    
    temp = m_smoke;
    m_smoke = m_smokeNext;
    m_smokeNext = temp;
}

void Simulation::render_texture(uint8_t *pixels) {
    d_render_texture<<<1000, 256>>>(
        m_pixels->m_deviceData,
        m_smoke->m_deviceData,
        m_u->m_deviceData,
        m_v->m_deviceData,
        m_obstacles->m_deviceData,
        m_width,
        m_height
    );
    m_pixels->from_device(pixels);
}
