#include "hip/hip_runtime.h"
#include "Simulation.cuh"
#include <iostream>
#include <iomanip>

__global__ void d_advect_vel(
    float *uField,
    float *vField,
    float *uNext,
    float *vNext,
    uint8_t *obstacles,
    unsigned int w,
    unsigned int h,
    float deltaT,
    float metersPerCell)
{
    int stride = gridDim.x * blockDim.x;
    int max_index = w * h;

    for (int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < max_index; i+=stride) {
        int x = i % w;
        int y = i / w;

        // Find the velocities at x, y
        float u = (uField[(y * (w + 1)) + (x    )] + uField[((y    ) * (w + 1)) + (x + 1)]) / 2.0;
        float v = (vField[(y * (w    )) + (x    )] + vField[((y + 1) * (w    )) + (x    )]) / 2.0;

        // Add velocity to left side of the screen
        if ((x > 0) && (x < 2) && (y < h) && (y > 0)) {
            uField[i] = 0.02;
        }

        // Calculate the coordinates of the sample location
        float newX = max(min(x - (u * deltaT / metersPerCell) - 0.5, (float) w), 0.0f);
        float newY = max(min(y - (v * deltaT / metersPerCell), (float) h), 0.0f);

        float xFrac = newX - (long)newX;
        float yFrac = newY - (long)newY;
        
        // Sample uField
        float w11 = (1 - xFrac) * (1 - yFrac);
        float w12 = (1 - xFrac) * (    yFrac);
        float w21 = (    xFrac) * (1 - yFrac);
        float w22 = (    xFrac) * (    yFrac);

        uNext[i] = (
            w11 * uField[(unsigned int)(((newY    ) * w) + (newX    ))] +
            w12 * uField[(unsigned int)(((newY + 1) * w) + (newX    ))] +
            w21 * uField[(unsigned int)(((newY    ) * w) + (newX + 1))] +
            w22 * uField[(unsigned int)(((newY + 1) * w) + (newX + 1))] 
        );

        // Sample vField
        newX = max(min(x - (u * deltaT / metersPerCell), (float) w), 0.0f);
        newY = max(min(y - (v * deltaT / metersPerCell) - 0.5, (float) h), 0.0f);

        xFrac = newX - (long)newX;
        yFrac = newY - (long)newY;

        w11 = (1 - xFrac) * (1 - yFrac);
        w12 = (1 - xFrac) * (    yFrac);
        w21 = (    xFrac) * (1 - yFrac);
        w22 = (    xFrac) * (    yFrac);

        vNext[i] = (
            w11 * vField[(unsigned int)(((newY    ) * w) + (newX    ))] +
            w12 * vField[(unsigned int)(((newY + 1) * w) + (newX    ))] +
            w21 * vField[(unsigned int)(((newY    ) * w) + (newX + 1))] +
            w22 * vField[(unsigned int)(((newY + 1) * w) + (newX + 1))] 
        );
    }
}

__global__ void d_advect_smoke(
    float *smoke,
    float *smokeNext,
    float *uField,
    float *vField,
    uint8_t *obstacles,
    unsigned int w,
    unsigned int h,
    float deltaT,
    float metersPerCell)
{
    int stride = gridDim.x * blockDim.x;
    int max_index = w * h;

    for (int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < max_index; i+=stride) {
        int x = i % w;
        int y = i / w;

        // Advect smoke
        // Find the velocities at x, y
        float u = (uField[(y * (w + 1)) + (x    )] + uField[((y    ) * (w + 1)) + (x + 1)]) / 2.0;
        float v = (vField[(y * (w    )) + (x    )] + vField[((y + 1) * (w    )) + (x    )]) / 2.0;

        // Add smoke to left side of the screen
        if ((x > 0) && (x < 2) && (y < h) && (y > 0)) {
            smoke[i] = 1.0;
        }

        // Calculate the coordinates of the sample location
        float newX = max(min(x - (u * deltaT / metersPerCell), (float) w), 0.0f);
        float newY = max(min(y - (v * deltaT / metersPerCell), (float) h), 0.0f);

        float xFrac = newX - (long)newX;
        float yFrac = newY - (long)newY;
        
        // Sample the smoke at location - delta_t * velocity
        float w11 = (1 - xFrac) * (1 - yFrac);
        float w12 = (1 - xFrac) * (    yFrac);
        float w21 = (    xFrac) * (1 - yFrac);
        float w22 = (    xFrac) * (    yFrac);

        smokeNext[i] = (
            w11 * smoke[(unsigned int)(((newY    ) * w) + (newX    ))] +
            w12 * smoke[(unsigned int)(((newY + 1) * w) + (newX    ))] +
            w21 * smoke[(unsigned int)(((newY    ) * w) + (newX + 1))] +
            w22 * smoke[(unsigned int)(((newY + 1) * w) + (newX + 1))] 
        );
    }
}

__global__ void d_render_texture(uint8_t *pixels, float *smoke, float *uField, float *vField, unsigned int width, unsigned int height) {
    int stride = gridDim.x * blockDim.x;
    int max_index = width * height;

    for (int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < max_index; i+=stride) {
        // Clamp the smoke value to 0-255
        //int pixel_value = static_cast<int>(uField[i * width/(width + 1)] * 25500 * 2);
        int pixel_value = static_cast<int>(smoke[i] * 255);
        if (pixel_value > 255) {
            pixel_value = 255;
        } else if (pixel_value < 0) {
            pixel_value = 0;
        }
        pixels[(4*i) + 1] = pixel_value;
    }
}

Simulation::Simulation(unsigned int width, unsigned int height, float dt) {
    m_width = width;
    m_height = height;
    m_u =         new GPUField<float>(  (width + 1) *  height     , 0.001);
    m_v =         new GPUField<float>(   width      * (height + 1), 0.00);
    m_uNext =     new GPUField<float>(  (width + 1) *  height     );
    m_vNext =     new GPUField<float>(   width      * (height + 1));
    m_smoke =     new GPUField<float>(   width      *  height     );
    m_smokeNext = new GPUField<float>(   width      *  height     );
    m_obstacles = new GPUField<uint8_t>( width      *  height     );
    m_pixels =    new GPUField<uint8_t>(4 * width   * height);
}

Simulation::~Simulation() {
    delete m_u;
    delete m_v;
    delete m_uNext;
    delete m_vNext;
    delete m_smoke;
    delete m_obstacles;
    delete m_pixels;
}

void Simulation::to_device(){
    m_u->to_device();
    m_v->to_device();
    m_uNext->to_device();
    m_vNext->to_device();
    m_smoke->to_device();
    m_smokeNext->to_device();
    m_obstacles->to_device();

    // d_pixels only goes from device to host
    //d_pixels.to_device();
}

void Simulation::from_device(){
    m_u->from_device();
    m_v->from_device();
    m_uNext->from_device();
    m_vNext->from_device();
    m_smoke->from_device();
    m_smokeNext->from_device();
    m_obstacles->from_device();
}

void Simulation::step() {
    //project();
    //advect_velocity();

    //m_u->m_hostData[0] = 0.11;
    //m_u->m_hostData[1] = 0.22;
    this->to_device();   
    //printf("u: %f\n", m_u->m_hostData[1]);

    d_advect_smoke<<<1000, 256>>>(
        m_smoke->m_deviceData,
        m_smokeNext->m_deviceData,
        m_u->m_deviceData,
        m_v->m_deviceData,
        m_obstacles->m_deviceData,
        m_width,
        m_height,
        0.0001,
        0.0001);

    d_advect_vel<<<1000, 256>>>(
        m_u->m_deviceData,
        m_v->m_deviceData,
        m_uNext->m_deviceData,
        m_vNext->m_deviceData,
        m_obstacles->m_deviceData,
        m_width,
        m_height,
        0.0001,
        0.0001);

    this->from_device();

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }

    GPUField<float> *temp;

    temp = m_u;
    m_u = m_uNext;
    m_uNext = temp;

    temp = m_v;
    m_v = m_vNext;
    m_vNext = temp;
    
    temp = m_smoke;
    m_smoke = m_smokeNext;
    m_smokeNext = temp;
}

void Simulation::render_texture(uint8_t *pixels) {
    d_render_texture<<<1000, 256>>>(m_pixels->m_deviceData, m_smoke->m_deviceData, m_u->m_deviceData, m_v->m_deviceData, m_width, m_height);
    m_pixels->from_device(pixels);
}
